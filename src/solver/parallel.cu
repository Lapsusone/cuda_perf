#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "parallel.cuh"

using std::cout;
using std::flush;
using std::endl;

__global__ void plus100Kernel(int *input, int* output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 100)
    {
        output[i] = input[i] + 100;
    }
}

void plus100(int n_block, int n_thread)
{
    int *d_input = 0;
    int *d_output = 0;
    hipMalloc((void**)&d_input, 100 * sizeof(int));
    hipMalloc((void**)&d_output, 100 * sizeof(int));
 
    srand(time(NULL)); 

    int* matrice = (int*)malloc(sizeof(int) * 100); 

    for(int i = 0; i < 100; i++)
    {
        matrice[i] = rand() % 100;
    }

    // Copier vers le dispositif
    hipMemcpy(d_input, matrice, 100 * sizeof(int), hipMemcpyHostToDevice);

    // Appeler le kernel avec 100 blocs
    plus100Kernel<<<n_block, n_thread>>>(d_input, d_output);

    // Attendre que le kernel ait fini, puis copier vers l'hôte
    hipDeviceSynchronize();
    hipMemcpy(matrice, d_output, 100 * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < 100; i++)
    {
        printf("%d\n", matrice[i]);
    }
} 

